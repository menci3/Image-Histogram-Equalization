#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

#define COLOR_CHANNELS 0

__global__ void copy_image(const unsigned char *imageIn, unsigned char *imageOut, const int width, const int height, const int cpp) {
    int gidx = blockDim.x * blockIdx.x + threadIdx.x;
    int gidy = blockDim.y * blockIdx.y + threadIdx.y;
    
    if (gidx == 0 & gidy == 0) {
        printf("DEVICE: START COPY\n");
    }
    
    for (int i = gidx; i < height; i += blockDim.x * gridDim.x) {
        for (int j = gidy; j < width; j += blockDim.y * gridDim.y) {
            for (int c = 0; c < cpp; c += 1) {
                imageOut[(i * width + j) * cpp + c] = imageIn[(i * width + j) * cpp + c];
            }
        }
    }

}

// RGB to YUV conversion device function
__device__ void rgb_to_yuv_cuda(int R, int G, int B, int *Y, int *U, int *V) {
    *Y = (int)roundf(0.299f * R + 0.587f * G + 0.114f * B);
    *U = (int)roundf(-0.168736f * R - 0.331264f * G + 0.5f * B) + 128;
    *V = (int)roundf(0.5f * R - 0.418688f * G - 0.081312f * B) + 128;
}

// CUDA kernel for RGB to YUV conversion
__global__ void rgbToYuvKernel(const unsigned char *image_in, unsigned char *image_out, int width, int height) {
    // Calculate pixel position based on thread and block indices
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    
    // Check if the thread corresponds to a valid pixel
    if (row < height && col < width) {
        // Calculate index in the 1D array
        int index = (row * width + col) * 3;
        
        // Get RGB values
        int R = image_in[index];
        int G = image_in[index + 1];
        int B = image_in[index + 2];
        
        // Convert to YUV
        int Y, U, V;
        rgb_to_yuv_cuda(R, G, B, &Y, &U, &V);
        
        // Store YUV values
        image_out[index] = Y;
        image_out[index + 1] = U;
        image_out[index + 2] = V;
    }
}

__global__ void computeHistogramKernel(const unsigned char *image, int *histogram, int width, int height) {
    __shared__ int partial_histogram[256];

    int threadId = threadIdx.x;
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // Initialize shared memory
    for (int i = threadId; i < 256; i += blockDim.x) {
        partial_histogram[i] = 0;
    }
    __syncthreads();

    int numPixels = width * height;

    for (int i = globalId; i < numPixels; i += stride) {
        unsigned char y_value = image[i * 3];

        atomicAdd(&partial_histogram[y_value], 1);
    }
    __syncthreads();

    // Merge partial histogram into global histogram
    for (int i = threadId; i < 256; i += blockDim.x) {
        atomicAdd(&histogram[i], partial_histogram[i]);
    }
}

__global__ void computeHistogramKernelSimple(const unsigned char *image, int *histogram, int width, int height) {
    int globalId = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    int numPixels = width * height;

    for (int i = globalId; i < numPixels; i += stride) {
        unsigned char y_value = image[i * 3];
        atomicAdd(&histogram[y_value], 1);
    }
}


__global__ void computeCumulativeHistogram(int *histogram, int *cdf) {
   extern __shared__ int temp[256]; // 256 entries, 1 block

    int tid = threadIdx.x;

    // Load input into shared memory
    if (tid < 256) {
        temp[tid] = histogram[tid];
    }
    __syncthreads();

    // Upsweep
    int offset = 1;
    for (int d = 256 >> 1; d > 0; d >>= 1) {
        __syncthreads();
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;
            temp[bi] += temp[ai];
        }
        offset *= 2;
    }

    // Set last element to 0 for exclusive scan
    if (tid == 0) {
        temp[255] = 0;
    }

    // Downsweep
    for (int d = 1; d < 256; d *= 2) {
        offset >>= 1;
        __syncthreads();
        if (tid < d) {
            int ai = offset * (2 * tid + 1) - 1;
            int bi = offset * (2 * tid + 2) - 1;

            int t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    // Write result to output CDF (inclusive scan)
    if (tid < 256) {
        // For inclusive scan: add the original value to the exclusive scan result
        cdf[tid] = temp[tid];
    }
}

__global__ void computeCumulativeHistogramSimple(const int *histogram, int *cdf) {
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        int sum = 0;
        for (int i = 0; i < 256; ++i) {
            sum += histogram[i];
            cdf[i] = sum;
        }
    }
}

__global__ void calculateLuminanceLookupKernel(int *histogram_cumulative, int *luminance_lookup_table, int pixels) {
    __shared__ int min_value;

    int tid = threadIdx.x;

    // Initialize min_value to INT_MAX
    if (tid == 0) {
        min_value = INT_MAX;
    }
    __syncthreads();

    // Each thread checks its value
    if (tid < 256 && histogram_cumulative[tid] > 0 && histogram_cumulative[tid] < min_value) {
        atomicMin(&min_value, histogram_cumulative[tid]);
    }
    __syncthreads();

    // Calculate new luminance values
    if (tid < 256) {
        // Apply histogram equalization formula
        luminance_lookup_table[tid] = floor(((float)(histogram_cumulative[tid] - min_value) / (pixels - min_value)) * 255);
    }
}

__global__ void applyLuminanceAndConvertToRgbKernel(unsigned char *image, const int *luminance_lookup_table, int width, int height) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (row < height && col < width) {
        int index = (row * width + col) * 3;

        // Get YUV values
        int Y = image[index];
        int U = image[index + 1];
        int V = image[index + 2];

        // Apply new luminance from lookup table
        Y = luminance_lookup_table[Y];

        // Convert back to RGB
        int R, G, B;
        U -= 128;
        V -= 128;

        // YUV to RGB conversion with correct coefficients
        R = (int)roundf(Y + 1.402f * V);
        G = (int)roundf(Y - 0.344136f * U - 0.714136f * V);
        B = (int)roundf(Y + 1.772f * U);

        // Clamp values to [0, 255]
        //R = max(0, min(255, R));
        //G = max(0, min(255, G));
        //B = max(0, min(255, B));

        R = R < 0 ? 0 : (R > 255 ? 255 : R);
        G = G < 0 ? 0 : (G > 255 ? 255 : G);
        B = B < 0 ? 0 : (B > 255 ? 255 : B);

        // Store RGB values
        image[index] = R;
        image[index + 1] = G;
        image[index + 2] = B;
    }
}

void rgb_to_yuv(int R, int G, int B, int *Y, int *U, int *V) {
    *Y = (int)roundf(0.299f * R + 0.587f * G + 0.114f * B);
    *U = (int)roundf(-0.168736f * R - 0.331264f * G + 0.5f * B) + 128;
    *V = (int)roundf(0.5f * R - 0.418688f * G - 0.081312f * B) + 128;
}

void yuv_to_rgb(int Y, int U, int V, int *R, int *G, int *B) {
    U -= 128;
    V -= 128;

    *R = (int)roundf(Y + 1.402f * V);
    *G = (int)roundf(Y - 0.344136f * U - 0.714136f * V);
    *B = (int)roundf(Y + 1.772f * U);

    // Ensuring that the values stay in range [0,255]
    *R = *R < 0 ? 0 : (*R > 255 ? 255 : *R);
    *G = *G < 0 ? 0 : (*G > 255 ? 255 : *G);
    *B = *B < 0 ? 0 : (*B > 255 ? 255 : *B);
}

void sequential(const unsigned char *image_in, unsigned char *image_out, int width, int height) {
    printf("Sequential execution time\n");
    // 1. Transform the image from RGB to YUV space
    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            int index = (i * width + j) * 3;

            int R, G, B;
            int Y, U, V;

            R = image_in[index];
            G = image_in[index + 1];
            B = image_in[index + 2];

            rgb_to_yuv(R, G, B, &Y, &U, &V);

            image_out[index] = Y;
            image_out[index + 1] = U;
            image_out[index + 2] = V;
        }
    }

    // 2. Compute the luminance histogram
    int histogram[256] = {0};
    int y_value;

    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            int index = (i * width + j) * 3;

            y_value = image_out[index];
            histogram[y_value]++;
        }
    }

    // 3. Calculate the cumulative histogram
    int histogram_cumulative[256] = {0};

    histogram_cumulative[0] = histogram[0];

    for (int i = 1; i < 256; i++){
        histogram_cumulative[i] = histogram_cumulative[i - 1] + histogram[i];
    }

    // 4. Calculate new pixel luminances from original luminances based on the histogram equalization formula
    int luminance[256] = {0};
    int min_cumulative = INT_MAX;

    for (int i = 0; i < 256; i++) {
        if (histogram_cumulative[i] < min_cumulative) {
            if (histogram_cumulative[i] != 0) {
                min_cumulative = histogram_cumulative[i];
            }
        }
    }

    int pixels = width * height;

    for (int i = 0; i < 256; i++){
        luminance[i] = floor(((float)(histogram_cumulative[i] - min_cumulative) / (pixels - min_cumulative)) * 255);
    }

    // 5. Assign new luminance to each pixel
    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            int index = (i * width + j) * 3;

            image_out[index] = luminance[image_out[index]];
        }
    }

    // 6. Convert the image back to RGB colour space
    for (int i = 0; i < height; i++){
        for (int j = 0; j < width; j++){
            int index = (i * width + j) * 3;

            int Y, U, V;
            int R, G, B;

            Y = image_out[index];
            U = image_out[index + 1];
            V = image_out[index + 2];

            yuv_to_rgb(Y, U, V, &R, &G, &B);

            image_out[index] = R;
            image_out[index + 1] = G;
            image_out[index + 2] = B;
        }
    }
}

void parallel(unsigned char *image_in, unsigned char *image_out, int width, int height, int blockSizeInput) {
    hipError_t error;
    int imageSize = width * height * 3 * sizeof(unsigned char);

    // Set up execution configuration
    dim3 blockSize(blockSizeInput, blockSizeInput);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    int histogramThreads = 256;
    int histogramBlocks = min(16, (width * height + histogramThreads - 1) / histogramThreads);

    // Allocate device memory
    unsigned char *d_image_in, *d_image_out;
    error = hipMalloc(&d_image_in, imageSize);
    if (error != hipSuccess) {
        printf("Error allocating device memory for input image: %s\n", hipGetErrorString(error));
        return;
    }

    error = hipMalloc(&d_image_out, imageSize);
    if (error != hipSuccess) {
        printf("Error allocating device memory for output image: %s\n", hipGetErrorString(error));
        hipFree(d_image_in);
        return;
    }

    // Copy input image to device
    hipMemcpy(d_image_in, image_in, imageSize, hipMemcpyHostToDevice);

    int *d_histogram;
    int* d_cdf;
    int *d_luminance_lookup_table;

    hipMalloc(&d_luminance_lookup_table, 256 * sizeof(int));
    hipMalloc(&d_cdf, 256 * sizeof(int));
    hipMalloc(&d_histogram, 256 * sizeof(int));
    hipMemset(d_histogram, 0, 256 * sizeof(int));

    // Launch RGB to YUV conversion kernel
    // Run once to warm up the GPU and not invalidate the measurement
    rgbToYuvKernel<<<gridSize, blockSize>>>(d_image_in, d_image_out, width, height);

    /*hipEvent_t start, stop;
    float milliseconds = 0;
    float total_milliseconds = 0;
    int iterations = 10;

    for (int i = 0; i < iterations; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        rgbToYuvKernel<<<gridSize, blockSize>>>(d_image_in, d_image_out, width, height);

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        // Print time
        hipEventElapsedTime(&milliseconds, start, stop);
        total_milliseconds += milliseconds;
    }

    printf("RGBtoYUV Block size %i, time: %0.3f milliseconds \n", blockSizeInput * blockSizeInput, total_milliseconds/iterations);
    total_milliseconds = 0; */

    // Check for kernel launch errors
    error = hipGetLastError();
    if (error != hipSuccess) {
        printf("Error launching RGB to YUV kernel: %s\n", hipGetErrorString(error));
        hipFree(image_in);
        hipFree(image_out);
        return;
    }

    // Make sure kernel execution is finished
    hipDeviceSynchronize();

    // TODO: Add the remaining steps of histogram equalization
    // 1. Compute luminance histogram

    /*for (int i = 0; i < iterations; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        computeHistogramKernelSimple<<<histogramBlocks, histogramThreads>>>(d_image_out, d_histogram, width, height);
        hipDeviceSynchronize();

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        // Print time
        hipEventElapsedTime(&milliseconds, start, stop);
        total_milliseconds += milliseconds;
    }

    printf("histogram Block size %i, time: %0.3f milliseconds \n", blockSizeInput * blockSizeInput, total_milliseconds/iterations);
    total_milliseconds = 0; */


    computeHistogramKernelSimple<<<histogramBlocks, histogramThreads>>>(d_image_out, d_histogram, width, height);
    hipDeviceSynchronize();

    // 2. Calculate cumulative histogram (Bleloch scan)
    //computeCumulativeHistogram<<<1, 256>>>(d_histogram, d_cdf);
    computeCumulativeHistogramSimple<<<1, 1>>>(d_histogram, d_cdf);
    hipDeviceSynchronize();

    // 3. Calculate new pixel luminances
    calculateLuminanceLookupKernel<<<1, 256>>>(d_cdf, d_luminance_lookup_table, width * height);
    hipDeviceSynchronize();

    // 4. Apply new luminance to each pixel
    // 5. Convert back from YUV to RGB

    /*for (int i = 0; i < iterations; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);

        applyLuminanceAndConvertToRgbKernel<<<gridSize, blockSize>>>(d_image_out, d_luminance_lookup_table, width, height);
        hipDeviceSynchronize();

        hipEventRecord(stop);

        hipEventSynchronize(stop);

        // Print time
        hipEventElapsedTime(&milliseconds, start, stop);
        total_milliseconds += milliseconds;
    }

    printf("YUVtoRGB Block size %i, time: %0.3f milliseconds \n", blockSizeInput * blockSizeInput, total_milliseconds/iterations);*/

    applyLuminanceAndConvertToRgbKernel<<<gridSize, blockSize>>>(d_image_out, d_luminance_lookup_table, width, height);
    hipDeviceSynchronize();

    // Copy result back to host
    error = hipMemcpy(image_out, d_image_out, imageSize, hipMemcpyDeviceToHost);
    if (error != hipSuccess) {
        printf("Error copying output image to host: %s\n", hipGetErrorString(error));
    }

    // Free device memory
    hipFree(d_image_in);
    hipFree(d_image_out);
    hipFree(d_histogram);
    hipFree(d_cdf);
    hipFree(d_luminance_lookup_table);
}

int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("USAGE: sample input_image output_image\n");
        exit(EXIT_FAILURE);
    }

    char szImage_in_name[255];
    char szImage_out_name[255];

    snprintf(szImage_in_name, 255, "%s", argv[1]);
    snprintf(szImage_out_name, 255, "%s", argv[2]);

    // Load image from file and allocate space for the output image
    int width, height, cpp;
    unsigned char *h_imageIn = stbi_load(szImage_in_name, &width, &height, &cpp, COLOR_CHANNELS);

    if (h_imageIn == NULL) {
        printf("Error reading loading image %s!\n", szImage_in_name);
        exit(EXIT_FAILURE);
    }
    printf("Loaded image %s of size %dx%d.\n", szImage_in_name, width, height);
    const size_t datasize = width * height * cpp * sizeof(unsigned char);
    unsigned char *h_imageOut = (unsigned char *)malloc(datasize);

    /*clock_t begin, end;
    float elapsed_ms;

    begin = clock();

    sequential(h_imageIn, h_imageOut, width, height);

    end = clock();
    elapsed_ms = ((float)(end - begin) / CLOCKS_PER_SEC) * 1000.0;

    printf("Sequential method time: %.3f milliseconds\n", elapsed_ms);
    */

    // Setup Thread organization
    //dim3 blockSize(16, 16);
    //dim3 gridSize((height-1)/blockSize.x+1,(width-1)/blockSize.y+1);
    //dim3 gridSize(1, 1);

    unsigned char *d_imageIn;
    unsigned char *d_imageOut;

    // Allocate memory on the device
    checkCudaErrors(hipMalloc(&d_imageIn, datasize));
    checkCudaErrors(hipMalloc(&d_imageOut, datasize));

    // Parallel CUDA implementation
    printf("Parallel execution time\n");

    hipEvent_t start, stop;
    float milliseconds = 0;
    float total_milliseconds = 0;
    int iterations = 100;

    for (int block = 8; block <= 32; block *= 2) {
        printf("Block size %i\n", block * block);
        for (int i = 0; i < iterations; i++) {
            hipEventCreate(&start);
            hipEventCreate(&stop);

            hipEventRecord(start);

            parallel(h_imageIn, h_imageOut, width, height, block);

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            // Print time
            hipEventElapsedTime(&milliseconds, start, stop);
            total_milliseconds += milliseconds;
        }

        printf("Block size %i, time: %0.3f milliseconds \n", block * block, total_milliseconds/iterations);
        total_milliseconds = 0;
    }

    // Write the output file
    char szImage_out_name_temp[255];
    strncpy(szImage_out_name_temp, szImage_out_name, 255);
    char *token = strtok(szImage_out_name_temp, ".");
    char *FileType = NULL;
    while (token != NULL) {
        FileType = token;
        token = strtok(NULL, ".");
    }

    if (!strcmp(FileType, "png"))
        stbi_write_png(szImage_out_name, width, height, cpp, h_imageOut, width * cpp);
    else if (!strcmp(FileType, "jpg"))
        stbi_write_jpg(szImage_out_name, width, height, cpp, h_imageOut, 100);
    else if (!strcmp(FileType, "bmp"))
        stbi_write_bmp(szImage_out_name, width, height, cpp, h_imageOut);
    else
        printf("Error: Unknown image format %s! Only png, bmp, or bmp supported.\n", FileType);

    // Free device memory
    //checkCudaErrors(hipFree(d_imageIn));
    //checkCudaErrors(hipFree(d_imageOut));

    // Clean-up events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free host memory
    free(h_imageIn);
    free(h_imageOut);

    return 0;
}
